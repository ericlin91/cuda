#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>


#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// Lab4: You can use any other block size you wish.
#define BLOCK_SIZE 256

// Lab4: Host Helper Functions (allocate your own data structure...)


// Lab4: Device Functions


// Lab4: Kernel Functions


__global__ void scan_workefficient(float *g_odata, float *g_idata, int n)
{
    // Dynamically allocated shared memory for scan kernels
    extern  __shared__  float temp[];

    int thid = threadIdx.x;

    int offset = 1;

    // Cache the computational window in shared memory
    temp[2*thid]   = g_idata[2*thid];
    temp[2*thid+1] = g_idata[2*thid+1];

    // build the sum in place up the tree
    for (int d = n>>1; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)      
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            temp[bi] += temp[ai];
        }

        offset *= 2;
    }

    // scan back down the tree

    // clear the last element
    if (thid == 0)
    {
        temp[n - 1] = 0;
    }   

    // traverse down the tree building the scan in place
    for (int d = 1; d < n; d *= 2)
    {
        offset >>= 1;
        __syncthreads();

        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            float t   = temp[ai];
            temp[ai]  = temp[bi];
            temp[bi] += t;
        }
    }

    __syncthreads();

    // write results to global memory
    g_odata[2*thid]   = temp[2*thid];
    g_odata[2*thid+1] = temp[2*thid+1];
}


// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{

	scan_workefficient<<<1,512,4096>>>(outArray, inArray,numElements);


}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
